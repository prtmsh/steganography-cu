#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <random>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <openssl/sha.h>
#include <sstream>
#include <iomanip>

// Structure to hold pixel position
struct PixelPosition {
    int y;
    int x;
};

// CUDA kernel for embedding bits into the LSB of the blue channel
__global__ void embedBitsKernel(unsigned char* image, int width, int height, 
                               const PixelPosition* positions, const unsigned char* data, 
                               int totalBits) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < totalBits) {
        int byteIndex = idx / 8;
        int bitPos = 7 - (idx % 8);  // MSB first
        
        int y = positions[idx].y;
        int x = positions[idx].x;
        int pixelIdx = (y * width + x) * 3;  // 3 channels (BGR)
        
        // Get the bit from the data
        unsigned char bit = (data[byteIndex] >> bitPos) & 1;
        
        // Clear the LSB of the blue channel and set it to the current bit
        image[pixelIdx] = (image[pixelIdx] & 0xFE) | bit;
    }
}

// CUDA kernel for extracting bits from the LSB of the blue channel
__global__ void extractBitsKernel(const unsigned char* image, int width, int height,
                                 const PixelPosition* positions, unsigned char* bits,
                                 int totalBits) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < totalBits) {
        int y = positions[idx].y;
        int x = positions[idx].x;
        int pixelIdx = (y * width + x) * 3;  // 3 channels (BGR)
        
        // Extract the LSB of the blue channel
        bits[idx] = image[pixelIdx] & 1;
    }
}

// Function to compute SHA-256 hash from border pixels
std::string computeBorderHash(const cv::Mat& image) {
    int height = image.rows;
    int width = image.cols;
    
    // Extract border pixels
    std::vector<unsigned char> borderPixels;
    
    // Top row
    for (int x = 0; x < width; x++) {
        cv::Vec3b pixel = image.at<cv::Vec3b>(0, x);
        borderPixels.push_back(pixel[0]);
        borderPixels.push_back(pixel[1]);
        borderPixels.push_back(pixel[2]);
    }
    
    // Bottom row
    for (int x = 0; x < width; x++) {
        cv::Vec3b pixel = image.at<cv::Vec3b>(height - 1, x);
        borderPixels.push_back(pixel[0]);
        borderPixels.push_back(pixel[1]);
        borderPixels.push_back(pixel[2]);
    }
    
    // Left column (excluding corners)
    for (int y = 1; y < height - 1; y++) {
        cv::Vec3b pixel = image.at<cv::Vec3b>(y, 0);
        borderPixels.push_back(pixel[0]);
        borderPixels.push_back(pixel[1]);
        borderPixels.push_back(pixel[2]);
    }
    
    // Right column (excluding corners)
    for (int y = 1; y < height - 1; y++) {
        cv::Vec3b pixel = image.at<cv::Vec3b>(y, width - 1);
        borderPixels.push_back(pixel[0]);
        borderPixels.push_back(pixel[1]);
        borderPixels.push_back(pixel[2]);
    }
    
    // Compute SHA-256 hash
    unsigned char hash[SHA256_DIGEST_LENGTH];
    SHA256_CTX sha256;
    SHA256_Init(&sha256);
    SHA256_Update(&sha256, borderPixels.data(), borderPixels.size());
    SHA256_Final(hash, &sha256);
    
    // Convert hash to hexadecimal string
    std::stringstream ss;
    for (int i = 0; i < SHA256_DIGEST_LENGTH; i++) {
        ss << std::hex << std::setw(2) << std::setfill('0') << static_cast<int>(hash[i]);
    }
    
    return ss.str();
}

// Function to generate pseudo-random positions based on hash
std::vector<PixelPosition> generatePseudoRandomPositions(const std::string& hashValue, int height, int width, int numBits) {
    // Convert hash to an integer seed
    unsigned long long seed = 0;
    std::istringstream iss(hashValue.substr(0, 16));
    iss >> std::hex >> seed;
    
    // Create a list of all interior pixel coordinates
    std::vector<PixelPosition> interiorPixels;
    for (int y = 1; y < height - 1; y++) {
        for (int x = 1; x < width - 1; x++) {
            interiorPixels.push_back({y, x});
        }
    }
    
    // Check if we have enough pixels
    int totalInteriorPixels = interiorPixels.size();
    if (totalInteriorPixels < numBits) {
        throw std::runtime_error("Image is too small for the message. Need at least " + 
                                std::to_string(numBits) + " interior pixels.");
    }
    
    // Shuffle based on the seed
    std::mt19937 gen(seed);
    std::shuffle(interiorPixels.begin(), interiorPixels.end(), gen);
    
    // Return needed positions
    return std::vector<PixelPosition>(interiorPixels.begin(), interiorPixels.begin() + numBits);
}

// Function to embed a message into an image
int embedMessage(const std::string& inputPath, const std::string& outputPath, const std::string& message) {
    // Load the image
    cv::Mat image = cv::imread(inputPath);
    if (image.empty()) {
        throw std::runtime_error("Could not open image at " + inputPath);
    }
    
    int height = image.rows;
    int width = image.cols;
    
    // Compute hash from border pixels
    std::string borderHash = computeBorderHash(image);
    
    // Convert message to bytes
    std::vector<unsigned char> messageBytes(message.begin(), message.end());
    int messageLength = messageBytes.size();
    
    // Ensure message length isn't too large
    if (messageLength > 65535) {
        throw std::runtime_error("Message too large - maximum size is 65535 bytes");
    }
    
    // Create a 2-byte header with the message length
    std::vector<unsigned char> lengthBytes = {
        static_cast<unsigned char>((messageLength >> 8) & 0xFF),  // High byte
        static_cast<unsigned char>(messageLength & 0xFF)          // Low byte
    };
    
    // Combine header and message
    std::vector<unsigned char> dataToHide;
    dataToHide.insert(dataToHide.end(), lengthBytes.begin(), lengthBytes.end());
    dataToHide.insert(dataToHide.end(), messageBytes.begin(), messageBytes.end());
    int totalBits = dataToHide.size() * 8;
    
    // Generate embedding positions
    std::vector<PixelPosition> positions = generatePseudoRandomPositions(borderHash, height, width, totalBits);
    
    // Allocate device memory
    unsigned char* d_image;
    PixelPosition* d_positions;
    unsigned char* d_data;
    
    // Flatten the image for GPU processing
    std::vector<unsigned char> flatImage;
    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            cv::Vec3b pixel = image.at<cv::Vec3b>(y, x);
            flatImage.push_back(pixel[0]);  // B
            flatImage.push_back(pixel[1]);  // G
            flatImage.push_back(pixel[2]);  // R
        }
    }
    
    // Allocate and copy data to GPU
    hipMalloc(&d_image, flatImage.size() * sizeof(unsigned char));
    hipMalloc(&d_positions, positions.size() * sizeof(PixelPosition));
    hipMalloc(&d_data, dataToHide.size() * sizeof(unsigned char));
    
    hipMemcpy(d_image, flatImage.data(), flatImage.size() * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_positions, positions.data(), positions.size() * sizeof(PixelPosition), hipMemcpyHostToDevice);
    hipMemcpy(d_data, dataToHide.data(), dataToHide.size() * sizeof(unsigned char), hipMemcpyHostToDevice);
    
    // Define block and grid sizes
    int blockSize = 256;
    int gridSize = (totalBits + blockSize - 1) / blockSize;
    
    // Launch kernel
    embedBitsKernel<<<gridSize, blockSize>>>(d_image, width, height, d_positions, d_data, totalBits);
    
    // Copy the result back to host
    hipMemcpy(flatImage.data(), d_image, flatImage.size() * sizeof(unsigned char), hipMemcpyDeviceToHost);
    
    // Update the image with embedded data
    int idx = 0;
    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            cv::Vec3b& pixel = image.at<cv::Vec3b>(y, x);
            pixel[0] = flatImage[idx++];  // B
            pixel[1] = flatImage[idx++];  // G
            pixel[2] = flatImage[idx++];  // R
        }
    }
    
    // Free GPU memory
    hipFree(d_image);
    hipFree(d_positions);
    hipFree(d_data);
    
    // Save the watermarked image (ensuring PNG format)
    std::string finalOutputPath = outputPath;
    if (finalOutputPath.substr(finalOutputPath.find_last_of(".") + 1) != "png") {
        finalOutputPath = finalOutputPath.substr(0, finalOutputPath.find_last_of(".")) + ".png";
    }
    
    std::vector<int> compressionParams;
    compressionParams.push_back(cv::IMWRITE_PNG_COMPRESSION);
    compressionParams.push_back(0);  // 0 = no compression
    
    cv::imwrite(finalOutputPath, image, compressionParams);
    
    std::cout << "DEBUG: Embedded message length: " << messageLength << " bytes" << std::endl;
    return messageLength * 8;
}

// Function to extract a message from a watermarked image
std::string extractMessage(const std::string& inputPath) {
    // Load the watermarked image
    cv::Mat image = cv::imread(inputPath);
    if (image.empty()) {
        throw std::runtime_error("Could not open image at " + inputPath);
    }
    
    int height = image.rows;
    int width = image.cols;
    
    // Compute hash from border pixels
    std::string borderHash = computeBorderHash(image);
    
    // First extract the 2-byte header (16 bits)
    int headerBits = 16;
    std::vector<PixelPosition> headerPositions = generatePseudoRandomPositions(borderHash, height, width, headerBits);
    
    // Flatten the image for GPU processing
    std::vector<unsigned char> flatImage;
    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            cv::Vec3b pixel = image.at<cv::Vec3b>(y, x);
            flatImage.push_back(pixel[0]);  // B
            flatImage.push_back(pixel[1]);  // G
            flatImage.push_back(pixel[2]);  // R
        }
    }
    
    // Allocate device memory
    unsigned char* d_image;
    PixelPosition* d_positions;
    unsigned char* d_bits;
    
    // Allocate and copy data to GPU
    hipMalloc(&d_image, flatImage.size() * sizeof(unsigned char));
    hipMalloc(&d_positions, headerPositions.size() * sizeof(PixelPosition));
    hipMalloc(&d_bits, headerBits * sizeof(unsigned char));
    
    hipMemcpy(d_image, flatImage.data(), flatImage.size() * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_positions, headerPositions.data(), headerPositions.size() * sizeof(PixelPosition), hipMemcpyHostToDevice);
    
    // Define block and grid sizes
    int blockSize = 256;
    int gridSize = (headerBits + blockSize - 1) / blockSize;
    
    // Launch kernel to extract header bits
    extractBitsKernel<<<gridSize, blockSize>>>(d_image, width, height, d_positions, d_bits, headerBits);
    
    // Copy the extracted bits back to host
    std::vector<unsigned char> headerBitsHost(headerBits);
    hipMemcpy(headerBitsHost.data(), d_bits, headerBits * sizeof(unsigned char), hipMemcpyDeviceToHost);
    
    // Convert bits to bytes (2 bytes for length)
    unsigned char lengthBytes[2] = {0, 0};
    for (int i = 0; i < 2; i++) {
        for (int j = 0; j < 8; j++) {
            // MSB first
            lengthBytes[i] = (lengthBytes[i] << 1) | headerBitsHost[i*8 + j];
        }
    }
    
    // Extract message length
    int messageLength = (lengthBytes[0] << 8) | lengthBytes[1];
    std::cout << "DEBUG: Detected message length: " << messageLength << " bytes" << std::endl;
    
    // Safety check
    if (messageLength > 65535 || messageLength <= 0) {
        return "Error: Invalid message length detected: " + std::to_string(messageLength);
    }
    
    // Generate positions for the full message
    int totalBits = headerBits + (messageLength * 8);
    std::vector<PixelPosition> allPositions = generatePseudoRandomPositions(borderHash, height, width, totalBits);
    
    // Only need positions for the message part
    std::vector<PixelPosition> messagePositions(allPositions.begin() + headerBits, allPositions.end());
    int messageBits = messageLength * 8;
    
    // Free previous GPU allocations
    hipFree(d_positions);
    hipFree(d_bits);
    
    // Allocate new memory for message extraction
    hipMalloc(&d_positions, messagePositions.size() * sizeof(PixelPosition));
    hipMalloc(&d_bits, messageBits * sizeof(unsigned char));
    
    hipMemcpy(d_positions, messagePositions.data(), messagePositions.size() * sizeof(PixelPosition), hipMemcpyHostToDevice);
    
    // Calculate new grid size for message bits
    gridSize = (messageBits + blockSize - 1) / blockSize;
    
    // Launch kernel to extract message bits
    extractBitsKernel<<<gridSize, blockSize>>>(d_image, width, height, d_positions, d_bits, messageBits);
    
    // Copy the extracted bits back to host
    std::vector<unsigned char> messageBitsHost(messageBits);
    hipMemcpy(messageBitsHost.data(), d_bits, messageBits * sizeof(unsigned char), hipMemcpyDeviceToHost);
    
    // Convert bits to bytes
    std::vector<unsigned char> messageBytes(messageLength);
    for (int i = 0; i < messageLength; i++) {
        unsigned char byte = 0;
        for (int j = 0; j < 8; j++) {
            // MSB first
            if (i*8 + j < messageBitsHost.size()) {
                byte = (byte << 1) | messageBitsHost[i*8 + j];
            }
        }
        messageBytes[i] = byte;
    }
    
    // Free GPU memory
    hipFree(d_image);
    hipFree(d_positions);
    hipFree(d_bits);
    
    // Convert bytes to string
    try {
        std::string message(messageBytes.begin(), messageBytes.end());
        return message;
    } catch (const std::exception& e) {
        // If decoding fails, return hex representation
        std::stringstream hexData;
        for (size_t i = 0; i < std::min(messageBytes.size(), size_t(32)); i++) {
            hexData << std::hex << std::setw(2) << std::setfill('0') << static_cast<int>(messageBytes[i]);
        }
        return "Error: Could not decode message. First 32 bytes in hex: " + hexData.str() + "...";
    }
}